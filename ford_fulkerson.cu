#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
using namespace std;

// A structure that stores the flow along a directed edge
struct edge{
	int u, v, c, f;
};

// finds augmenting path in non-deterministic fashion
__global__ void find_augmenting_path(edge *d_edges, int m, int *vis, int *par, 
									 int *current_flow, int *progress){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < m){
		int u = d_edges[id].u, v = d_edges[id].v, c = d_edges[id].c, f = d_edges[id].f;
		// checking if forward edge uv exists in residual graph
		if(vis[u] && !vis[v] && f < c && atomicCAS(par+v, -1, id) == -1){
			vis[v] = 1;
			current_flow[v] = min(current_flow[u], c - f);
			atomicAdd(progress, 1);
		}
		// checking if reverse edge vu exists in residual graph
		if(vis[v] && !vis[u] && f > 0 && atomicCAS(par+u, -1, id) == -1){
			vis[u] = 1;
			current_flow[u] = min(current_flow[v], f);
			atomicAdd(progress, 1);
		}
	}

}

// augemnts along path found by find_augmenting_path
__global__ void augment(edge* d_edges, int* par, int t, int flow){
	int cur = t;
	while(cur){
		int idx = par[cur];
		int u = d_edges[idx].u, v = d_edges[idx].v;
		if(cur == u){
			d_edges[idx].f -= flow;
			cur = v;
		}
		else{
			d_edges[idx].f += flow;
			cur = u;
		}
	}
}

int main(int argc, char* argv[]){
	auto clk=clock();

	if(argc < 2){
		cout<<"Enter file name"<<endl;
		return 0;
	}

	int n, m, INF = 1000000000;
	edge *edges, *d_edges;
	int *vis, *par, *progress, *current_flow;	

	ifstream fin(argv[1]);
	fin >> n >> m;

	edges = new edge[m];
	for(int i = 0; i < m; i++){
		fin >> edges[i].u >> edges[i].v >> edges[i].c;
		edges[i].u--;
		edges[i].v--;
		edges[i].f = 0;
	}	
	
	hipMalloc(&d_edges, m * sizeof(edge));
	hipMalloc(&vis, n * sizeof(int));
	hipMalloc(&par, n * sizeof(int));
	hipMalloc(&current_flow, n * sizeof(int));
	hipMalloc(&progress, sizeof(int));

	hipMemcpy(d_edges, edges, m*sizeof(edge), hipMemcpyHostToDevice);

	int threads = 1024;
	int blocks = ceil((float)m/threads);
	int total_flow = 0;

	while(true){

		hipMemset(vis, 0, n * sizeof(int));
		hipMemset(par, -1, n * sizeof(int));
		hipMemset(current_flow, 0, n * sizeof(int));
		hipMemset(vis, 1, sizeof(int));

		hipMemcpy(current_flow, &INF, sizeof(int), hipMemcpyHostToDevice);

		int prog, t_reachable, cur_flow;
		
		// this loop performs search for augmenting path in parallel fashion
		// loop breaks when there is no new vertex that is reached in the last iteration
		do{
			hipMemset(progress, 0, sizeof(int));
			find_augmenting_path<<<blocks,threads>>>(d_edges, m, vis, par, current_flow, progress);
			hipMemcpy(&prog, progress, sizeof(int), hipMemcpyDeviceToHost);
		
		}while(prog);

		hipMemcpy(&t_reachable, vis + n - 1, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&cur_flow, current_flow + n - 1, sizeof(int), hipMemcpyDeviceToHost);
		
		if(!t_reachable){
			assert(!cur_flow);
			break;
		}
		
		// has to be done serially
		augment<<<1,1>>>(d_edges, par, n-1 , cur_flow);
		
		total_flow += cur_flow;
	}

	double t_elapsed = (double)(clock()-clk)/CLOCKS_PER_SEC;
	printf("|V|:%d |E|:%d Flow:%d\nTime:%f\n", n, m, total_flow, t_elapsed);	
}